
#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <iomanip>

using namespace std;
typedef long long ll;


__global__ void kernel(double* a, double* b, double* answer, ll n)
{
    ll offset = gridDim.x * blockDim.x;
    ll idx = blockDim.x * blockIdx.x + threadIdx.x;
    for(ll i = idx; i < n; i = i + offset){
        answer[i] = a[i] - b[i];
    }
    return;
}


int main()
{
    ios_base::sync_with_stdio(false);
    cin.tie(NULL);

    ll n;
    cin >> n;

    double* a = new double[n];
    double* b = new double[n];
    double* answer = new double[n];

    for (ll i = 0; i < n; ++i) {
        cin >> a[i];
    }
    for (ll i = 0; i < n; ++i) {
        cin >> b[i];
    }

    double* nums1;
    double* nums2;
    double* res;

    if (hipMalloc((void**)&nums1, sizeof(double) * n) != hipSuccess) {
        cerr << "ERROR!   46\n";
        abort();
    }
    if (hipMalloc((void**)&nums2, sizeof(double) * n) != hipSuccess) {
        cerr << "ERROR!   50\n";
        abort();
    }
    if (hipMalloc((void**)&res, sizeof(double) * n) != hipSuccess) {
        cerr << "ERROR!   54\n";
        abort();
    }

    if (hipMemcpy(nums1, a, sizeof(double) * n, hipMemcpyHostToDevice) != hipSuccess) {
        cerr << "ERROR!   59\n";
        abort();
    }
    if (hipMemcpy(nums2, b, sizeof(double) * n, hipMemcpyHostToDevice) != hipSuccess) {
        cerr << "ERROR!   63\n";
        abort();
    }

    kernel <<<256, 256>>> (nums1, nums2, res, n);

    if (hipGetLastError() != hipSuccess) {
        cerr << "ERROR!   70\n";
        abort();
    }

    if (hipMemcpy(answer, res, sizeof(double) * n, hipMemcpyDeviceToHost) != hipSuccess) {
        cerr << "ERROR!   75\n";
        abort();
    }

    cout.precision(10);
    cout.setf(ios::scientific);
    for (ll i = 0; i < n; ++i) {
        cout << answer[i] << " ";
    }
    cout << "\n";

    if (hipFree(nums1) != hipSuccess) {
        cerr << "ERROR!   87\n";
        abort();
    }
    if (hipFree(nums2) != hipSuccess) {
        cerr << "ERROR!   91\n";
        abort();
    }
    if (hipFree(res) != hipSuccess) {
        cerr << "ERROR!   95\n";
        abort();
    }

    delete[] a;
    delete[] b;
    delete[] answer;

    return 0;
}
