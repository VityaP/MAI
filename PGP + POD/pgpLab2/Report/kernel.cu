#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4* dst, int w, int h, int wNew, int hNew)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int y = idy; y < hNew; y += offsety) {
        for (int x = idx; x < wNew; x += offsetx) {
            int i = (float)(x + 0.5) * w / wNew - 0.5;
            int j = (float)(y + 0.5) * h / hNew - 0.5;
            float xx = (float)(x + 0.5) * w / wNew - 0.5 - i;
            float yy = (float)(y + 0.5) * h / hNew - 0.5 - j;

            if (xx < (float)0.0) {
                i -= 1;
                xx += (float)1.0;
            }

            if (yy < (float)0.0) {
                j -= 1;
                yy += (float)1.0;
            }
            uchar4 pIJ = tex2D(tex, i, j);
            uchar4 pI1J = tex2D(tex, i + 1, j);
            uchar4 pIJ1 = tex2D(tex, i, j + 1);
            uchar4 pI1J1 = tex2D(tex, i + 1, j + 1);
            //uchar4 res;
            float r = pIJ.x * (1.0f - xx) * (1.0f - yy) + pI1J.x * xx * (1.0f - yy) + pIJ1.x * (1.0f - xx) * yy + pI1J1.x * xx * yy;
            float g = pIJ.y * (1.0f - xx) * (1.0f - yy) + pI1J.y * xx * (1.0f - yy) + pIJ1.y * (1.0f - xx) * yy + pI1J1.y * xx * yy;
            float b = pIJ.z * (1.0f - xx) * (1.0f - yy) + pI1J.z * xx * (1.0f - yy) + pIJ1.z * (1.0f - xx) * yy + pI1J1.z * xx * yy;
            float w = pIJ.w;
            dst[y * wNew + x] = make_uchar4(r, g, b, w);
        }
    }
    return;
}


int main()
{
    string input;
    string output;
    int wNew, hNew;
    cin >> input >> output >> wNew >> hNew;

    uchar4* data = nullptr;
    int w, h;

    ifstream inputFile(input, std::ios::in | std::ios::binary);
    if (inputFile.is_open()) {
        if (!inputFile.read((char*)&w, sizeof(w))) {
            cerr << "ERROR: can't read from file " << __LINE__ << endl;
            abort();
        }
        if (!inputFile.read((char*)&h, sizeof(h))) {
            cerr << "ERROR: can't read from file " << __LINE__ << endl;
            abort();
        }
        int size = w * h;
        if (size < wNew * hNew) {
            size = wNew * hNew;
        }
        data = new uchar4[size];
        if (!inputFile.read((char*)data, w * h * sizeof(uchar4))) {
            cerr << "ERROR: can't read from file " << __LINE__ << endl;
            abort();
        }
        inputFile.close();
    }
    else {
        cerr << "ERROR: can't open file " << __LINE__ << endl;
        abort();
    }

    hipArray* arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));


    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;


    CSC(hipBindTextureToArray(tex, arr, ch));

    uchar4* new_image;
    CSC(hipMalloc(&new_image, sizeof(uchar4) * hNew * wNew));


    kernel <<<dim3(32, 32), dim3(32, 32) >>> (new_image, w, h, wNew, hNew);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, new_image, sizeof(uchar4) * hNew * wNew, hipMemcpyDeviceToHost));

    std::ofstream outputFile(output, std::ios::out | std::ios::binary);
    if (outputFile.is_open()) {
        if (!outputFile.write((char*)&wNew, sizeof(wNew))) {
            cerr << "ERROR: can't open write " << __LINE__ << endl;
            abort();
        }
        if (!outputFile.write((char*)&hNew, sizeof(hNew))) {
            cerr << "ERROR: can't open write " << __LINE__ << endl;
            abort();
        }
        if (!outputFile.write((char*)data, wNew * hNew * sizeof(uchar4))) {
            cerr << "ERROR: can't open write " << __LINE__ << endl;
            abort();
        }
        outputFile.close();
    }
    else {
        cerr << "ERROR: can't open file " << __LINE__ << endl;
        abort();
    }

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(new_image));

    delete[] data;
    return 0;
}
