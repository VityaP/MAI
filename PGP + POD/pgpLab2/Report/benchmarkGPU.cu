#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4* dst, int w, int h, int wNew, int hNew)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int y = idy; y < hNew; y += offsety) {
        for (int x = idx; x < wNew; x += offsetx) {
            int i = (float)(x + 0.5) * w / wNew - 0.5;
            int j = (float)(y + 0.5) * h / hNew - 0.5;
            float xx = (float)(x + 0.5) * w / wNew - 0.5 - i;
            float yy = (float)(y + 0.5) * h / hNew - 0.5 - j;

            if (xx < (float)0.0) {
                i -= 1;
                xx += (float)1.0;
            }

            if (yy < (float)0.0) {
                j -= 1;
                yy += (float)1.0;
            }
            uchar4 pIJ = tex2D(tex, i, j);
            uchar4 pI1J = tex2D(tex, i + 1, j);
            uchar4 pIJ1 = tex2D(tex, i, j + 1);
            uchar4 pI1J1 = tex2D(tex, i + 1, j + 1);
            //uchar4 res;
            float r = pIJ.x * (1.0f - xx) * (1.0f - yy) + pI1J.x * xx * (1.0f - yy) + pIJ1.x * (1.0f - xx) * yy + pI1J1.x * xx * yy;
            float g = pIJ.y * (1.0f - xx) * (1.0f - yy) + pI1J.y * xx * (1.0f - yy) + pIJ1.y * (1.0f - xx) * yy + pI1J1.y * xx * yy;
            float b = pIJ.z * (1.0f - xx) * (1.0f - yy) + pI1J.z * xx * (1.0f - yy) + pIJ1.z * (1.0f - xx) * yy + pI1J1.z * xx * yy;
            float w = pIJ.w;
            dst[y * wNew + x] = make_uchar4(r, g, b, w);
        }
    }
    return;
}


int main()
{

    int w, h;
    int wNew, hNew;
    cin >> w >> h >> wNew >> hNew;

    uchar4* data = nullptr;
    int size = w * h;
    if (size < wNew * hNew) {
        size = wNew * hNew;
    }
    data = new uchar4[size];

    hipArray* arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));


    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;


    CSC(hipBindTextureToArray(tex, arr, ch));

    uchar4* new_image;
    CSC(hipMalloc(&new_image, sizeof(uchar4) * hNew * wNew));


    hipEvent_t start, stop;
    cudaErrorCheck(hipEventCreate(&start));
    cudaErrorCheck(hipEventCreate(&stop));
    cudaErrorCheck(hipEventRecord(start, 0));

    kernel <<<dim3(16, 16), dim3(16, 16) >>> (new_image, w, h, wNew, hNew);
    CSC(hipGetLastError());

    cudaErrorCheck(hipEventRecord(stop, 0));
    cudaErrorCheck(hipEventSynchronize(stop));

    double time = -1.0;
    cudaErrorCheck(hipEventElapsedTime(&time, start, stop));
    cudaErrorCheck(hipEventDestroy(start));
    cudaErrorCheck(hipEventDestroy(stop));
    cout << time << endl;



    CSC(hipMemcpy(data, new_image, sizeof(uchar4) * hNew * wNew, hipMemcpyDeviceToHost));

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(new_image));

    delete[] data;
    return 0;
}
