#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;
	for(y = idy; y < h; y += offsety)
		for(x = idx; x < w; x += offsetx) {
			p = tex2D(tex, x, y);
			out[y * w + x] = make_uchar4(~p.x, ~p.y, ~p.z, p.w);
		}
}

int main() {
	int w, h;
	FILE *fp = fopen("in.data", "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	// Подготовка данных для текстуры
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));
	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

	// Подготовка текстурной ссылки, настройка интерфейса работы с данными
	tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;		// Без интерполяции при обращении по дробным координатам
	tex.normalized = false;						// Режим нормализации координат: без нормализации

	// Связываем интерфейс с данными
	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

	kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_out, w, h);
	CSC(hipGetLastError());

	CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	// Отвязываем данные от текстурной ссылки
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen("out.data", "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
