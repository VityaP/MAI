
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  					\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)


__global__ void kernel(int *arr, int n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while(idx < n) {
		arr[idx] *= 2;
		idx += offset;
	}
}

int main() {
	int i, n = 100000000;
	int *arr = (int *)malloc(sizeof(int) * n);
	for(i = 0; i < n; i++)
		arr[i] = i;

	int *dev_arr;

	CSC(hipMalloc(&dev_arr, sizeof(int) * n));
	CSC(hipMemcpy(dev_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	kernel<<<256, 256>>>(dev_arr, n);
	CSC(hipGetLastError());

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));

	printf("time = %f\n", t);

	CSC(hipMemcpy(arr, dev_arr, sizeof(int) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(dev_arr));

	for(i = n - 10; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
	free(arr);
	return 0;
}
