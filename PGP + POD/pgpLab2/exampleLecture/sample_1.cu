
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(float *arr, int n) {
	int i, idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
	int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
	for(i = idx; i < n; i += offset)	
		arr[i] *= arr[i];
}

int main() {
	int i, n = 100000;
	float *arr = (float *)malloc(sizeof(float) * n);
	for(i = 0; i < n; i++)
		arr[i] = i;

	float *dev_arr;
	hipMalloc(&dev_arr, sizeof(float) * n);
	hipMemcpy(dev_arr, arr, sizeof(float) * n, hipMemcpyHostToDevice);

	kernel<<<256, 256>>>(dev_arr, n);

	hipMemcpy(arr, dev_arr, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipFree(dev_arr);
	for(i = 0; i < 100; i++)
		printf("%f ", arr[i]);
	printf("\n");
	free(arr);
	return 0;
}
