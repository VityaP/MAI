// #include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "hip/hip_runtime.h"

// using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

#define ASCENDING 1
#define DESCENDING 0

__device__ void compASC(int* left, int* right){
    if( *left > *right ){
        int tmp = *left;
        *left = *right;
        *right = tmp;
    }
}

__device__ void compDESC(int* left, int* right){
    if( *left < *right ){
        int tmp = *left;
        *left = *right;
        *right = tmp;
    }
}

#define ThreadPerBlock 512
// !!!Atention Victor!!!
// SharedSize should be equal to 2 * ThreadPerBlock
#define SharedSize 1024

#define _idxShared(id) ((id) + ((id) / 31))
#define _sizeShared(id) ((id) + ((id) / 31) + 100)
// 0   1   2   3   4   ... 29  30  X
// 31  32  33  34  35  ... 60  61  X
// 62  63  64  65  66  ... 91  92  X
// 93  94  95  96  97  ... 122 123 X
// 124 125 126 127 128 ...


__global__ void BitonicSortShared(int* nums, int n, int iterShift, int iterToStart){

    __shared__ int shared[_sizeShared(SharedSize)];

    int idx = blockIdx.x * SharedSize;
    int offset = SharedSize * gridDim.x;
    int prev = idx - (idx & (1 << iterShift));
    int j;
    int iter, localStep;
    int comp = ASCENDING;

    for(int i = idx; i < n; i += offset){
        if ((i >> iterShift) != (prev >> iterShift)){
            comp ^= 1;
        }
        prev = i;

        shared[_idxShared(threadIdx.x)] = nums[i + threadIdx.x];
        shared[_idxShared(ThreadPerBlock + threadIdx.x)] = nums[i + ThreadPerBlock + threadIdx.x];
        __syncthreads();

        for (iter = 2 * iterToStart; iter > 1; iter /= 2){
            for (localStep = iter / 2; localStep > 0; localStep /= 2){

                j = (2 * threadIdx.x) - (threadIdx.x & (localStep - 1));
                if( comp == ASCENDING ){
                    compASC(shared + _idxShared(j), shared + _idxShared(localStep + j));
                }
                else{
                    compDESC(shared + _idxShared(j), shared + _idxShared(localStep + j));
                }
                __syncthreads();

            }
        }

        nums[i + threadIdx.x] = shared[_idxShared(threadIdx.x)];
        nums[i + ThreadPerBlock + threadIdx.x] = shared[_idxShared(ThreadPerBlock + threadIdx.x)];
    }
}

__global__ void BitonicSort(int* nums, int n, int iterShift, int localStep, int localShift){

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = blockDim.x * gridDim.x;

    int prev;
    int comp = ASCENDING;
    int i = 2 * idx - (idx & (localStep - 1));
    int kernelShift = (localStep > offset) ? localStep : offset;

    if( (i >> iterShift) % 2 == 1){
        comp = DESCENDING;
    }

    while(i < n){
        if( comp == ASCENDING ){
            compASC(nums + i, nums + i + localStep);
        }
        else{
            compDESC(nums + i, nums + i + localStep);
        }

        prev = i;
        i += offset;

        if( (i >> localShift) != (prev >> localShift) ){
            i += kernelShift;
            if( (i >> iterShift) != (prev >> iterShift) ){
                comp ^= 1;
            }
        }
    }
}

int main(){

    int i, n;
    int powerBound = 1;
    fread(&n, sizeof(int), 1, stdin);

    while(powerBound < n){
        powerBound *= 2;
    }

    int* nums = (int*) malloc(powerBound * sizeof(int));
    fread(nums, sizeof(int), n, stdin);
    for(i = n; i < powerBound; ++i){
        nums[i] = INT_MAX;
    }

    // cout << "\n====================\n";
    // for(int i = 0; i < powerBound; ++i){
    //     cout << nums[i] << " ";
    // }
    // cout << "\n====================\n";

    int* device_nums;
    CSC(hipMalloc( (void **) &device_nums, powerBound * sizeof(int)));
    CSC(hipMemcpy(device_nums, nums, powerBound * sizeof(int), hipMemcpyHostToDevice));

    int iterShift = 1;
    int iteration, localShift, localStep;

    for(iteration = 2; iteration <= powerBound; iteration *= 2){
        localShift = iterShift - 1;
        for(localStep = iteration / 2; localStep > 0; localStep /= 2){

            if (localStep == SharedSize / 2){
                BitonicSortShared<<<16, ThreadPerBlock>>>(device_nums, powerBound, iterShift, localStep);
                CSC(hipGetLastError());
                break;
            }
            else{
                BitonicSort<<<64, 256>>>(device_nums, powerBound, iterShift, localStep, localShift);
                CSC(hipGetLastError());
            }
            
            localShift -= 1;
        }

        iterShift += 1;
    }

    CSC(hipMemcpy(nums, device_nums, powerBound * sizeof(int), hipMemcpyDeviceToHost));
    CSC(hipFree(device_nums));

    // cout << "\n====================\n";
    // for(int i = 0; i < powerBound; ++i){
    //     cout << nums[i] << " ";
    // }
    // cout << "\n====================\n";

    fwrite(nums, sizeof(int), n, stdout);
    free(nums);

    return 0;
}