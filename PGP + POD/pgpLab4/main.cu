#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cmath>
#include <stdbool.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

class Comparator{
public:
    __host__ __device__ bool operator()(const double a, const double b) const{
        return fabs(a) < fabs(b);
    }
};

__global__ void swapLines(double * matrix, double* identity, int n, int i, int j) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    int k;
    double tmp;
    for (k = idx; k < n; k += offset) {
        tmp = matrix[k * n + i];
        matrix[k * n + i] = matrix[k * n + j];
        matrix[k * n + j] = tmp;

        tmp = identity[k * n + i];
        identity[k * n + i] = identity[k * n + j];
        identity[k * n + j] = tmp;
    }
}

__global__ void devideIdentity(double* matrix, double* identity, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;
    int i, j;
    for (i = idx; i < n; i += offsetx) {
        for (j = idy; j < n; j += offsety) {
            identity[j * n + i] /= matrix[i * n + i];
        }
    }
}

__global__ void makeDownNull(double* matrix, double* identity, int n, int x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;

    int i, j;
    double particion;
    for (i = x + 1 + idx; i < n; i += offsetx) {
        particion = -matrix[x * n + i] / matrix[x * n + x];
        for (j = x + 1 + idy; j < n; j += offsety) {
            // matrix[j * n + i] = fma(-matrix[x * n + i] / matrix[x * n + x], matrix[j * n + x], matrix[j * n + i]);
            matrix[j * n + i] = particion * matrix[j * n + x] + matrix[j * n + i];
        }
        for (j = idy; j < n; j += offsety) {
            // identity[j * n + i] = fma(-matrix[x * n + i] / matrix[x * n + x], identity[j * n + x], identity[j * n + i]);
            identity[j * n + i] = particion * identity[j * n + x] + identity[j * n + i];
        }
    }

}

__global__ void makeUpNull(double* matrix, double* identity, int n, int x) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;

    int i, j;
    double particion;
    for (i = x - 1 - idx; i >= 0; i -= offsetx) {
        particion = -matrix[x * n + i] / matrix[x * n + x];
        for (j = idy; j < n; j += offsety) {
            // identity[j * n + i] = fma(-matrix[x * n + i] / matrix[x * n + x], identity[j * n + x], identity[j * n + i]);
            identity[j * n + i] = particion * identity[j * n + x] + identity[j * n + i];
        }
    }
}


int main() {

    std::ios_base::sync_with_stdio(false);
	std::cin.tie(NULL);

    int n;
    cin >> n;
    if( n <= 0 ){
        return 0;
    }

    int i, j;
    double *matrix = (double*) malloc( n * n * sizeof(double) );
    double *identity = (double*) malloc( n * n * sizeof(double) );

    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            cin >> matrix[j * n + i];
        }
    }

    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            if( i != j){
                identity[i * n + j] = 0.0;
            }
            else{
                identity[i * n + j] = 1.0;
            }
        }
    }


    
    double* dev_matrix;
    double* dev_identity;
    hipMalloc(&dev_matrix, sizeof(double) * n * n);
    hipMalloc(&dev_identity, sizeof(double) * n * n);
    hipMemcpy(dev_matrix, matrix, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_identity, identity, sizeof(double) * n * n, hipMemcpyHostToDevice);
    const thrust::device_ptr<double> ptr = thrust::device_pointer_cast(dev_matrix);

    const Comparator comp;
    dim3 block(32, 16);
    dim3 thread(32, 16);

    for (i = 0; i < n - 1; ++i) {
        const int max_idx = thrust::max_element(ptr + i * n + i, ptr + (i + 1) * n, comp) - ptr - i * n;
        if (max_idx != i){
            swapLines<<<256, 256>>>(dev_matrix, dev_identity, n, i, max_idx);
        }
        makeDownNull<<<block, thread>>>(dev_matrix, dev_identity, n, i);
    }


    for (i = n - 1; i > 0; i--) {
        // const int max_idx = thrust::max_element(ptr + i * n + i, ptr + (i + 1) * n, compare) - ptr - i * n;
        // if (max_idx != i){
        //     swapLines<<<256, 256>>>(dev_matrix, dev_identity, n, i, max_idx);
        // }
        makeUpNull<<<block, thread>>>(dev_matrix, dev_identity, n, i);
    }

    devideIdentity<<<block, thread>>>(dev_matrix, dev_identity, n);


    hipMemcpy(matrix, dev_matrix, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(identity, dev_identity, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipFree(dev_matrix);
    hipFree(dev_identity);

    cout << scientific;
    cout.precision(10);
    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            cout << identity[j * n + i] << " ";
        }
        cout << "\n";
    }
    
    delete[] matrix;
    delete[] identity;

    return 0;
}