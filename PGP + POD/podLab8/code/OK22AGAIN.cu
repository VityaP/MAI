#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "mpi.h"
#include <string>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

// Индексация внутри блока
#define _i(i, j, k) ( ((k) + 1) * (blockY + 2) * (blockX + 2) + ((j) + 1) * (blockX + 2) + (i) + 1 )
#define _ix(id) ( ( (id) % (blockX + 2) ) - 1)
#define _iy(id) ( ( ( (id) % ((blockY + 2) * (blockX + 2)) ) / (blockX + 2) ) - 1)
#define _iz(id) ( ( (id) / ((blockY + 2) * (blockX + 2)) ) - 1)

// Индексация по блокам (процессам)
#define _ib(i, j, k) ( (k) * (gridX * gridY) + (j) * gridX + (i) )
#define _ibx(id) (( (id) % (gridX * gridY) ) % gridX)
#define _iby(id) (( (id) % (gridX * gridY) ) / gridX)
#define _ibz(id) (  (id) / (gridX * gridY) )


__global__ void kernel_copy_xy(double *plane, double *data, int blockX, int blockY, int blockZ, int k, bool direction, double defVal){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
	int i, j;

    if( direction == true ){
		for(j = idy; j < blockY; j += offsety){
			for(i = idx; i < blockX; i += offsetx){
                plane[j * blockX + i] = data[_i(i, j, k)];
            }
		}
	}
	else{
		if( plane != NULL ){
			for(j = idy; j < blockY; j += offsety){
				for(i = idx; i < blockX; i += offsetx){
					data[_i(i, j, k)] = plane[j * blockX + i];
				}
			}
		}
		else{
			for(j = idy; j < blockY; j += offsety){
				for(i = idx; i < blockX; i += offsetx){
					data[_i(i, j, k)] = defVal;
				}
			}
		}
	}
}


__global__ void kernel_copy_yz(double *plane, double *data, int blockX, int blockY, int blockZ, int i, bool direction, double defVal){
    int idy = blockIdx.x * blockDim.x + threadIdx.x;
    int idz = blockIdx.y * blockDim.y + threadIdx.y;
    int offsety = blockDim.x * gridDim.x;
    int offsetz = blockDim.y * gridDim.y;
	int j, k;

    if( direction == true ){
		for(k = idz; k < blockZ; k += offsetz){
			for(j = idy; j < blockY; j += offsety){
                plane[k * blockY + j] = data[_i(i, j, k)];
            }
		}
	}
	else{
		if( plane != NULL ){
			for(k = idz; k < blockZ; k += offsetz){
				for(j = idy; j < blockY; j += offsety){
					data[_i(i, j, k)] = plane[k * blockY + j];
				}
			}
		}
		else{
			for(k = idz; k < blockZ; k += offsetz){
				for(j = idy; j < blockY; j += offsety){
					data[_i(i, j, k)] = defVal;
				}
			}
		}
	}
}


__global__ void kernel_copy_xz(double *plane, double *data, int blockX, int blockY, int blockZ, int j, bool direction, double defVal){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idz = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsetz = blockDim.y * gridDim.y;
	int i, k;

    if( direction == true ){
		for(k = idz; k < blockZ; k += offsetz){
			for(i = idx; i < blockX; i += offsetx){
                plane[k * blockX + i] = data[_i(i, j, k)];
            }
		}
	}
	else{
		if( plane != NULL ){
			for(k = idz; k < blockZ; k += offsetz){
				for(i = idx; i < blockX; i += offsetx){
					data[_i(i, j, k)] = plane[k * blockX + i];
				}
			}
		}
		else{
			for(k = idz; k < blockZ; k += offsetz){
				for(i = idx; i < blockX; i += offsetx){
					data[_i(i, j, k)] = defVal;
				}
			}
		}
	}
}


__global__ void kernel1(double* next, double* data, int blockX, int blockY, int blockZ, double hx, double hy, double hz){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int offsetz = blockDim.z * gridDim.z;
    int i, j, k;
    for(i = idx; i < blockX; i += offsetx ){
        for(j = idy; j < blockY; j += offsety ){
            for(k = idz; k < blockZ; k += offsetz ){
				next[_i(i, j, k)] = 0.5 * ( (data[_i(i + 1, j, k)] + data[_i(i - 1, j, k)]) / (hx * hx) +
											(data[_i(i, j + 1, k)] + data[_i(i, j - 1, k)]) / (hy * hy) +
											(data[_i(i, j, k + 1)] + data[_i(i, j, k - 1)]) / (hz * hz)
										) / 
										(1.0 / (hx * hx) + 1.0 / (hy * hy) + 1.0 / (hz * hz));
            }
        }
    }

}


__global__ void kernel2(double* next, double* data, int blockX, int blockY, int blockZ){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int offsetz = blockDim.z * gridDim.z;
    int i, j, k;
    for(i = idx - 1; i <= blockX; i += offsetx ){
        for(j = idy - 1; j <= blockY; j += offsety ){
            for(k = idz - 1; k <= blockZ; k += offsetz ){
				if( (i != -1) && (j != -1) && (k != -1) && (i != blockX) && (j != blockY) && (k != blockZ) ){
					data[_i(i, j, k)] = abs(next[_i(i, j, k)] - data[_i(i, j, k)]);
				}
				else{
					data[_i(i, j, k)] = 0.0;
				}
            }
        }
    }

}


int main(int argc, char *argv[]) {

	double downU, upU, leftU, rightU, frontU, backU;
	int gridX, gridY, gridZ;
    int blockX, blockY, blockZ;
    
	int stringLen;
	string outputFile;
    double epsilon;

	double lx, ly, lz;
    double startU;

	int id, ib, jb, kb;
	int i, j, k, step;
	int numproc;
	double *data, *temp, *next;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	// hipDeviceProp_t devProp;
	// CSC(hipGetDeviceProperties(&devProp, 0));
	int deviceCount;
	CSC(hipGetDeviceCount(&deviceCount));
	CSC(hipSetDevice(id % deviceCount));
	
	if (id == 0) {

		cin >> gridX >> gridY >> gridZ;
		cin >> blockX >> blockY >> blockZ;
		cin >> outputFile;
		cin >> epsilon;
		cin >> lx >> ly >> lz;
		cin >> downU >> upU >> leftU >> rightU >> frontU >> backU;
		cin >> startU;
		stringLen = outputFile.length();

		// cerr << gridX << " " << gridY << " " << gridZ << "\n";
		// cerr << blockX << " " << blockY << " " << blockZ << "\n";
		// cerr << outputFile << "\n";
		// cerr << epsilon << "\n";
		// cerr << lx << " " << ly << " " << lz << "\n";
		// cerr << downU << " " << upU << " " << leftU << " ";
		// cerr << rightU << " " << frontU << " " << backU << "\n";
		// cerr << startU << "\n";
	}

	MPI_Bcast(&blockX, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&blockY, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&blockZ, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&gridX, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&gridY, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&gridZ, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lz, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&epsilon, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&downU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&upU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&leftU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&rightU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&frontU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&backU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&startU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&stringLen, 1, MPI_INT, 0, MPI_COMM_WORLD);

	if( id != 0){
		outputFile.resize(stringLen);
	}

	char x;
	for(i = 0; i < stringLen; ++i){
		if (id == 0) {
			x = outputFile[i];
		}
		MPI_Bcast(&x, 1, MPI_CHAR, 0, MPI_COMM_WORLD);
		outputFile[i] = x;
	}
	
	ib = _ibx(id);
	jb = _iby(id);
	kb = _ibz(id);
	
	double hx = lx / (double) (gridX * blockX);	
	double hy = ly / (double) (gridY * blockY);
	double hz = lz / (double) (gridZ * blockZ);
	
	int sizeOfBuff = blockX * blockY;
	sizeOfBuff = max(sizeOfBuff, blockX * blockZ);
	sizeOfBuff = max(sizeOfBuff, blockY * blockZ);
	// infoBuffer = (double *)malloc(sizeof(double) * sizeOfBuff);
	int buffer_size = 12 * sizeOfBuff * sizeof(double) + 12 * MPI_BSEND_OVERHEAD;	// Выделение буфера для буферезированной отправки сообщений
	double *buffer = (double *)malloc(buffer_size);
	MPI_Buffer_attach(buffer, buffer_size);

	data = (double *)malloc(sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2));	
	next = (double *)malloc(sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2));

	for(i = -1; i <= blockX; ++i){					// Инициализация блока
		for(j = -1; j <= blockY; ++j){
			for(k = -1; k <= blockZ; ++k){
				data[_i(i, j, k)] = startU;
			}
		}
	}

	double* dev_data;
	CSC(hipMalloc( (void **) &dev_data, sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2)));
	CSC(hipMemcpy(dev_data, data, sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2), hipMemcpyHostToDevice));
	
	double* dev_next;
	CSC(hipMalloc( (void **) &dev_next, sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2)));
	
	double* plane_xy_CPU = (double *)malloc(sizeof(double) * blockX * blockY);
	double* plane_xy;
	CSC(hipMalloc( (void **) &plane_xy, sizeof(double) * blockX * blockY));

	double* plane_yz_CPU = (double *)malloc(sizeof(double) * blockY * blockZ);
	double* plane_yz;
	CSC(hipMalloc( (void **) &plane_yz, sizeof(double) * blockY * blockZ));
	
	double* plane_xz_CPU = (double *)malloc(sizeof(double) * blockX * blockZ);
	double* plane_xz;
	CSC(hipMalloc( (void **) &plane_xz, sizeof(double) * blockX * blockZ));

	dim3 blocks(32, 32);
	dim3 threads(32, 32);

	bool work = true;
	double localMax = 0.0;
	double* globalMaxes = (double *) malloc(sizeof(double) * numproc);
	int tmpSize1 = blockY * blockZ;
	int tmpSize2 = blockX * blockZ;
	int tmpSize3 = blockX * blockY;


	while(work == true) {

		// MPI_Barrier(MPI_COMM_WORLD);

		localMax = 0.0;

		if (ib + 1 < gridX) {					
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		infoBuffer[j * blockZ + k] = data[_i(blockX - 1, j, k)];
			// 	}
			// }
			kernel_copy_yz<<<blocks, threads>>>(plane_yz, dev_data, blockX, blockY, blockZ, blockX - 1, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_yz_CPU, plane_yz, sizeof(double) * blockY * blockZ, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_yz_CPU, tmpSize1, MPI_DOUBLE, _ib(ib + 1, jb, kb), id, MPI_COMM_WORLD);
		}

		if (jb + 1 < gridY) {
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		infoBuffer[i * blockZ + k] = data[_i(i, blockY - 1, k)];
			// 	}
			// }
			kernel_copy_xz<<<blocks, threads>>>(plane_xz, dev_data, blockX, blockY, blockZ, blockY - 1, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_xz_CPU, plane_xz, sizeof(double) * blockX * blockZ, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_xz_CPU, tmpSize2, MPI_DOUBLE, _ib(ib, jb + 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb + 1 < gridZ) {
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		infoBuffer[i * blockY + j] = data[_i(i, j, blockZ - 1)];
			// 	}
			// }
			kernel_copy_xy<<<blocks, threads>>>(plane_xy, dev_data, blockX, blockY, blockZ, blockZ - 1, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_xy_CPU, plane_xy, sizeof(double) * blockX * blockY, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_xy_CPU, tmpSize3, MPI_DOUBLE, _ib(ib, jb, kb + 1), id, MPI_COMM_WORLD);
		}
	
		if (ib > 0) {
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		infoBuffer[j * blockZ + k] = data[_i(0, j, k)];
			// 	}
			// }
			kernel_copy_yz<<<blocks, threads>>>(plane_yz, dev_data, blockX, blockY, blockZ, 0, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_yz_CPU, plane_yz, sizeof(double) * blockY * blockZ, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_yz_CPU, tmpSize1, MPI_DOUBLE, _ib(ib - 1, jb, kb), id, MPI_COMM_WORLD);
		}
		
		if (jb > 0) {
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		infoBuffer[i * blockZ + k] = data[_i(i, 0, k)];
			// 	}
			// }
			kernel_copy_xz<<<blocks, threads>>>(plane_xz, dev_data, blockX, blockY, blockZ, 0, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_xz_CPU, plane_xz, sizeof(double) * blockX * blockZ, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_xz_CPU, tmpSize2, MPI_DOUBLE, _ib(ib, jb - 1, kb), id, MPI_COMM_WORLD);
		}

		if (kb > 0) {
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		infoBuffer[i * blockY + j] = data[_i(i, j, 0)];
			// 	}
			// }
			kernel_copy_xy<<<blocks, threads>>>(plane_xy, dev_data, blockX, blockY, blockZ, 0, true, 1.0);
			CSC(hipGetLastError());
			CSC(hipMemcpy(plane_xy_CPU, plane_xy, sizeof(double) * blockX * blockY, hipMemcpyDeviceToHost));
			MPI_Bsend(plane_xy_CPU, tmpSize3, MPI_DOUBLE, _ib(ib, jb, kb - 1), id, MPI_COMM_WORLD);
		}

		// Прием данных

		if (ib > 0) {
			MPI_Recv(plane_yz_CPU, tmpSize1, MPI_DOUBLE, _ib(ib - 1, jb, kb), _ib(ib - 1, jb, kb), MPI_COMM_WORLD, &status);
			CSC(hipMemcpy(plane_yz, plane_yz_CPU, sizeof(double) * tmpSize1, hipMemcpyHostToDevice));
			kernel_copy_yz<<<blocks, threads>>>(plane_yz, dev_data, blockX, blockY, blockZ, -1, false, 1.0);
			CSC(hipGetLastError());
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(-1, j, k)] = infoBuffer[j * blockZ + k];
			// 	}
			// }
		}
		else {
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(-1, j, k)] = leftU;
			// 	}
			// }
			kernel_copy_yz<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, -1, false, leftU);
			CSC(hipGetLastError());
		}

		if (jb > 0) {
			MPI_Recv(plane_xz_CPU, tmpSize2, MPI_DOUBLE, _ib(ib, jb - 1, kb), _ib(ib, jb - 1, kb), MPI_COMM_WORLD, &status);
			hipMemcpy(plane_xz, plane_xz_CPU, sizeof(double) * tmpSize2, hipMemcpyHostToDevice);
			kernel_copy_xz<<<blocks, threads>>>(plane_xz, dev_data, blockX, blockY, blockZ, -1, false, 1.0);
			CSC(hipGetLastError());
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(i, -1, k)] = infoBuffer[i * blockZ + k];
			// 	}
			// }
		}
		else {
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(i, -1, k)] = frontU;
			// 	}
			// }
			kernel_copy_xz<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, -1, false, frontU);
			CSC(hipGetLastError());
		}

		if (kb > 0) {
			MPI_Recv(plane_xy_CPU, tmpSize3, MPI_DOUBLE, _ib(ib, jb, kb - 1), _ib(ib, jb, kb - 1), MPI_COMM_WORLD, &status);
			hipMemcpy(plane_xy, plane_xy_CPU, sizeof(double) * tmpSize3, hipMemcpyHostToDevice);
			kernel_copy_xy<<<blocks, threads>>>(plane_xy, dev_data, blockX, blockY, blockZ, -1, false, 1.0);
			CSC(hipGetLastError());
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		data[_i(i, j, -1)] = infoBuffer[i * blockY + j];
			// 	}
			// }
		}
		else {
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		data[_i(i, j, -1)] = downU;
			// 	}
			// }
			kernel_copy_xy<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, -1, false, downU);
			CSC(hipGetLastError());
		}

		if (ib + 1 < gridX) {
			MPI_Recv(plane_yz_CPU, tmpSize1, MPI_DOUBLE, _ib(ib + 1, jb, kb), _ib(ib + 1, jb, kb), MPI_COMM_WORLD, &status);
			hipMemcpy(plane_yz, plane_yz_CPU, sizeof(double) * tmpSize1, hipMemcpyHostToDevice);
			kernel_copy_yz<<<blocks, threads>>>(plane_yz, dev_data, blockX, blockY, blockZ, blockX, false, 1.0);
			CSC(hipGetLastError());
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(blockX, j, k)] = infoBuffer[j * blockZ + k];
			// 	}
			// }
		}
		else {
			// for(j = 0; j < blockY; ++j){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(blockX, j, k)] = rightU;
			// 	}
			// }
			kernel_copy_yz<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, blockX, false, rightU);
			CSC(hipGetLastError());
		}

		if (jb + 1 < gridY) {
			MPI_Recv(plane_xz_CPU, tmpSize2, MPI_DOUBLE, _ib(ib, jb + 1, kb), _ib(ib, jb + 1, kb), MPI_COMM_WORLD, &status);
			hipMemcpy(plane_xz, plane_xz_CPU, sizeof(double) * tmpSize2, hipMemcpyHostToDevice);
			kernel_copy_xz<<<blocks, threads>>>(plane_xz, dev_data, blockX, blockY, blockZ, blockY, false, 1.0);
			CSC(hipGetLastError());
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(i, blockY, k)] = infoBuffer[i * blockZ + k];
			// 	}
			// }
		}
		else {
			// for(i = 0; i < blockX; ++i){
			// 	for(k = 0; k < blockZ; ++k){
			// 		data[_i(i, blockY, k)] = backU;
			// 	}
			// }
			kernel_copy_xz<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, blockY, false, backU);
			CSC(hipGetLastError());
		}

		if (kb + 1 < gridZ) {
			MPI_Recv(plane_xy_CPU, tmpSize3, MPI_DOUBLE, _ib(ib, jb, kb + 1), _ib(ib, jb, kb + 1), MPI_COMM_WORLD, &status);
			hipMemcpy(plane_xy, plane_xy_CPU, sizeof(double) * tmpSize3, hipMemcpyHostToDevice);
			kernel_copy_xy<<<blocks, threads>>>(plane_xy, dev_data, blockX, blockY, blockZ, blockZ, false, 1.0);
			CSC(hipGetLastError());
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		data[_i(i, j, blockZ)] = infoBuffer[i * blockY + j];
			// 	}
			// }
		}
		else {
			// for(i = 0; i < blockX; ++i){
			// 	for(j = 0; j < blockY; ++j){
			// 		data[_i(i, j, blockZ)] = upU;
			// 	}
			// }
			kernel_copy_xy<<<blocks, threads>>>(NULL, dev_data, blockX, blockY, blockZ, blockZ, false, upU);
			CSC(hipGetLastError());
		}

		// MPI_Barrier(MPI_COMM_WORLD);

		// Перевычисление значений температуры

		// for(i = 0; i < blockX; ++i){
		// 	for(j = 0; j < blockY; ++j){
		// 		for(k = 0; k < blockZ; ++k){
		// 			next[_i(i, j, k)] = 0.5 * ( (data[_i(i + 1, j, k)] + data[_i(i - 1, j, k)]) * localConst1 +
		// 										(data[_i(i, j + 1, k)] + data[_i(i, j - 1, k)]) * localConst2 +
		// 										(data[_i(i, j, k + 1)] + data[_i(i, j, k - 1)]) * localConst3
		// 									  ) / divisor;
		// 			if( abs(next[_i(i, j, k)] - data[_i(i, j, k)]) > localMax ){
		// 				localMax = abs(next[_i(i, j, k)] - data[_i(i, j, k)]);
		// 			}
		// 		}
		// 	}
		// }

		kernel1<<<dim3(4, 4, 4), dim3(16, 8, 4)>>>(dev_next, dev_data, blockX, blockY, blockZ, hx, hy, hz);
		CSC(hipGetLastError());
		kernel2<<<dim3(4, 4, 4), dim3(16, 8, 4)>>>(dev_next, dev_data, blockX, blockY, blockZ);
		CSC(hipGetLastError());

		thrust::device_ptr<double> p_arr = thrust::device_pointer_cast(dev_data);
		thrust::device_ptr<double> resOfThrust = thrust::max_element(p_arr, p_arr + (blockX + 2) * (blockY + 2) * (blockZ + 2) );
		localMax = *resOfThrust;

		temp = dev_data;
		dev_data = dev_next;
		dev_next = temp;

		globalMaxes[id] = localMax;
		// MPI_Barrier(MPI_COMM_WORLD);
		MPI_Allgather(&localMax, 1, MPI_DOUBLE, globalMaxes, 1, MPI_DOUBLE, MPI_COMM_WORLD);
		
		work = false;
		for(step = 0; step < numproc; ++step ){
			if( globalMaxes[step] >= epsilon ){
				work = true;
			}
		}
		// if( work == false ){
		// 	cout << " " << work << " \n";
		// }
	}

	CSC(hipMemcpy(data, dev_data, sizeof(double) * (blockX + 2) * (blockY + 2) * (blockZ + 2), hipMemcpyDeviceToHost));
	CSC(hipFree(dev_data));
	CSC(hipFree(dev_next));
	CSC(hipFree(plane_xy));
	CSC(hipFree(plane_yz));
	CSC(hipFree(plane_xz));

	// MPI_Barrier(MPI_COMM_WORLD);
	int n_size = 20;
	char * buff = (char *) malloc(sizeof(char) * (blockX) * (blockY) * (blockZ) * n_size);
	memset(buff, ' ', (blockX) * (blockY) * (blockZ) * n_size * sizeof(char));
	
	for(k = 0; k < blockZ; ++k){
		for(j = 0; j < blockY; ++j) {
			for(i = 0; i < blockX; ++i){
				sprintf(buff + ( k * blockX * blockY + j * blockX + i ) * n_size, " %.6e ", data[_i(i, j, k)]);
			}
			if (_ibx(id) + 1 == gridX){
				buff[ (k * blockX * blockY + (j + 1) * blockX) * n_size - 1] = '\n';
			}
		}
	}
	for(i = 0; i < (blockX) * (blockY) * (blockZ) * n_size; ++i){
		if (buff[i] == '\0'){
			buff[i] = ' ';
		}
	}

	MPI_Datatype filetype;
	int array_of_sizes[3] = { gridZ * blockZ, gridY * blockY, gridX * blockX * n_size};
	int array_of_subsizes[3] = { blockZ, blockY, blockX * n_size};
	int array_of_starts[3] = {_ibz(id) * blockZ,_iby(id) * blockY, _ibx(id) * blockX * n_size};
	MPI_Type_create_subarray(3, array_of_sizes, array_of_subsizes, array_of_starts, MPI_ORDER_C , MPI_CHAR, &filetype);
	MPI_Type_commit(&filetype);

	MPI_File fp;
	MPI_File_delete(outputFile.c_str(), MPI_INFO_NULL);
	MPI_File_open(MPI_COMM_WORLD, outputFile.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);
	MPI_File_set_view(fp, 0, MPI_CHAR, filetype, "native", MPI_INFO_NULL);
	MPI_File_write_all(fp, buff, (blockX) * (blockY) * (blockZ) * n_size, MPI_CHAR, MPI_STATUS_IGNORE);
	MPI_File_close(&fp);


	// MPI_Barrier(MPI_COMM_WORLD);
	MPI_Buffer_detach(buffer, &buffer_size);
	MPI_Finalize();	

	free(buff);
	// free(infoBuffer);
	free(data);
	free(next);
	free(buffer);
	free(globalMaxes);

	return 0;
}