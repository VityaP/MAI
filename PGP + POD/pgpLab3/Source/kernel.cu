#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)


struct Image{
	int w;
	int h;
	uchar4* pixels;

	Image() {
		w = 0;
		h = 0;
		pixels = nullptr;
	}

	uchar4 GetPixel(int x, int y) {
		return pixels[x + y * w];
	}

	void ReadFromFile(string &input){

		int width;
		int height;
		ifstream inputFile(input, std::ios::in | std::ios::binary);

		if (inputFile.is_open()) {
			if (!inputFile.read((char*)&width, sizeof(width))) {
				cerr << "ERROR: can't read from file " << __LINE__ << endl;
				abort();
			}
			if (!inputFile.read((char*)&height, sizeof(height))) {
				cerr << "ERROR: can't read from file " << __LINE__ << endl;
				abort();
			}
			
			w = width;
			h = height;
			//pixels = (uchar4*)malloc(sizeof(uchar4) * width * height);
			pixels = new uchar4[width * height];

			if (!inputFile.read((char*)pixels, width * height * sizeof(uchar4))) {
				cerr << "ERROR: can't read from file " << __LINE__ << endl;
				abort();
			}
			inputFile.close();
		}
		else {
			cerr << "ERROR: can't open file " << __LINE__ << endl;
			abort();
		}

	}

	void WriteToFile(string &output){

		std::ofstream outputFile(output, std::ios::out | std::ios::binary);

		if (outputFile.is_open()) {
			if (!outputFile.write((char*)&w, sizeof(w))) {
				cerr << "ERROR: can't open write " << __LINE__ << endl;
				abort();
			}
			if (!outputFile.write((char*)&h, sizeof(h))) {
				cerr << "ERROR: can't open write " << __LINE__ << endl;
				abort();
			}
			if (!outputFile.write((char*)pixels, w * h * sizeof(uchar4))) {
				cerr << "ERROR: can't open write " << __LINE__ << endl;
				abort();
			}
			outputFile.close();
		}
		else {
			cerr << "ERROR: can't open file " << __LINE__ << endl;
			abort();
		}
	}

	void Delete() {
		delete[] pixels;
		w = 0;
		h = 0;
		pixels = nullptr;
	}
};



__constant__ double g_avgs[32][3];
__constant__ double g_inv_covs[32][3][3];


__device__ double classifierFunc(uchar4* pixel, int classInd){

	double diff[3];
	double res[3];

	diff[0] = (double) pixel->x - g_avgs[classInd][0];
	diff[1] = (double) pixel->y - g_avgs[classInd][1];
	diff[2] = (double) pixel->z - g_avgs[classInd][2];

	res[0] = 0.0;
	res[1] = 0.0;
	res[2] = 0.0;

	for (int j = 0; j < 3; ++j) {
		for (int k = 0; k < 3; ++k) {
			res[j] += diff[k] * g_inv_covs[classInd][k][j];
		}
	}

	double answer = 0.0;

	for (int k = 0; k < 3; ++k) {
		answer += res[k] * diff[k];
	}
	answer *= -1.0;

	return answer;
}


__global__ void kernel(uchar4* pixels, int size, int PGP_NC) {

	int idX = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetX = gridDim.x * blockDim.x;

	for (int i = idX; i < size; i = i + offsetX) {
		uchar4* pixel = &pixels[i];
		double maxVal = classifierFunc(pixel, 0);
		int maxInd = 0;

		for (int c = 1; c < PGP_NC; ++c){

			double tmpVal = classifierFunc(pixel, c);

			if (tmpVal > maxVal){
				maxVal = tmpVal;
				maxInd = c;
			}
		}

		pixel->w = (unsigned char) maxInd;
	}

}


int main(){

	string input;
	string output;
	Image image;

	cin >> input >> output;
	
	int nc;
	cin >> nc;
	vector< vector< pair<int, int> > > groups(nc);
	double avgs[32][3];
	double covs[32][3][3];
	double covs_inv[32][3][3];

	for (int i = 0; i < 32; ++i) {
		for (int j = 0; j < 3; ++j) {
			avgs[i][j] = 0.0;
		}
	}

	for (int i = 0; i < 32; ++i) {
		for (int j = 0; j < 3; ++j) {
			for (int k = 0; k < 3; ++k) {
				covs[i][j][k] = 0.0;
				covs_inv[i][j][k] = 0.0;
			}
		}
	}

	for (int i = 0; i < nc; ++i){
		int np;
		cin >> np;
		groups[i].resize(np);
		for (int j = 0; j < np; ++j){
			cin >> groups[i][j].first >> groups[i][j].second;
		}
	}

	image.ReadFromFile(input);

	for (int i = 0; i < nc; ++i) {
		
		//makeZeros(avgs[i]);
		//makeZeros(covs[i]);

		for (int j = 0; j < groups[i].size(); ++j){
			uchar4 pixel = image.GetPixel(groups[i][j].first, groups[i][j].second);
			avgs[i][0] += pixel.x;
			avgs[i][1] += pixel.y;
			avgs[i][2] += pixel.z;
		}

		avgs[i][0] /= (double) groups[i].size();
		avgs[i][1] /= (double) groups[i].size();
		avgs[i][2] /= (double) groups[i].size();

		for (int j = 0; j < groups[i].size(); ++j){

			uchar4 pixel = image.GetPixel(groups[i][j].first, groups[i][j].second);
			double diff[3];
			double productT[3][3];

			diff[0] = (double) pixel.x - avgs[i][0];
			diff[1] = (double) pixel.y - avgs[i][1];
			diff[2] = (double) pixel.z - avgs[i][2];

			for (int s1 = 0; s1 < 3; ++s1) {
				for (int s2 = 0; s2 < 3; ++s2) {
					productT[s1][s2] = diff[s1] * diff[s2];
				}
			}

			for (int s1 = 0; s1 < 3; ++s1) {
				for (int s2 = 0; s2 < 3; ++s2) {
					covs[i][s1][s2] += productT[s1][s2];
				}
			}

		}

		//Can't divide by 0
		//What if groups[i].size == 1
		//if (groups[i].size > 1) {
		double tmp = (double) groups[i].size() - 1.0;
		for (int s1 = 0; s1 < 3; ++s1) {
			for (int s2 = 0; s2 < 3; ++s2) {
				covs[i][s1][s2] /= tmp;
			}
		}
		//}

		double det = 0.0;
		det += covs[i][0][0] * covs[i][1][1] * covs[i][2][2];
		det += covs[i][0][1] * covs[i][1][2] * covs[i][2][0];
		det += covs[i][1][0] * covs[i][2][1] * covs[i][0][2];
		det -= covs[i][0][2] * covs[i][1][1] * covs[i][2][0];
		det -= covs[i][0][0] * covs[i][1][2] * covs[i][2][1];
		det -= covs[i][0][1] * covs[i][1][0] * covs[i][2][2];
		
		
		covs_inv[i][0][0] = (covs[i][1][1] * covs[i][2][2] - covs[i][2][1] * covs[i][1][2]) / det;
		covs_inv[i][0][1] = (covs[i][0][2] * covs[i][2][1] - covs[i][0][1] * covs[i][2][2]) / det;
		covs_inv[i][0][2] = (covs[i][0][1] * covs[i][1][2] - covs[i][0][2] * covs[i][1][1]) / det;
		covs_inv[i][1][0] = (covs[i][1][2] * covs[i][2][0] - covs[i][1][0] * covs[i][2][2]) / det;
		covs_inv[i][1][1] = (covs[i][0][0] * covs[i][2][2] - covs[i][0][2] * covs[i][2][0]) / det;
		covs_inv[i][1][2] = (covs[i][1][0] * covs[i][0][2] - covs[i][0][0] * covs[i][1][2]) / det;
		covs_inv[i][2][0] = (covs[i][1][0] * covs[i][2][1] - covs[i][2][0] * covs[i][1][1]) / det;
		covs_inv[i][2][1] = (covs[i][2][0] * covs[i][0][1] - covs[i][0][0] * covs[i][2][1]) / det;
		covs_inv[i][2][2] = (covs[i][0][0] * covs[i][1][1] - covs[i][1][0] * covs[i][0][1]) / det;
		
	}

	CSC(hipMemcpyToSymbol(HIP_SYMBOL(g_avgs), avgs, sizeof(double) * 32 * 3));
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(g_inv_covs), covs_inv, sizeof(double) * 32 * 9));


	uchar4* result;
	CSC(hipMalloc( (void **) &result, image.w * image.h * sizeof(uchar4)));
	CSC(hipMemcpy(result, image.pixels, image.w * image.h * sizeof(uchar4), hipMemcpyHostToDevice));

	int sz = image.w * image.h;
	kernel <<< 256, 256 >>> (result, sz, nc);
	CSC(hipGetLastError());

	CSC(hipMemcpy(image.pixels, result, image.w * image.h * sizeof(uchar4), hipMemcpyDeviceToHost));
	image.WriteToFile(output);

	image.Delete();
	CSC(hipFree(result));

	return 0;
}
