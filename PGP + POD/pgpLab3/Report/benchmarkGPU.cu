﻿#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <set>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)


__constant__ double g_avgs[32][3];
__constant__ double g_inv_covs[32][3][3];


__device__ double classifierFunc(uchar4* pixel, int classInd){

	double diff[3];
	double res[3];

	diff[0] = (double) pixel->x - g_avgs[classInd][0];
	diff[1] = (double) pixel->y - g_avgs[classInd][1];
	diff[2] = (double) pixel->z - g_avgs[classInd][2];

	res[0] = 0.0;
	res[1] = 0.0;
	res[2] = 0.0;

	for (int j = 0; j < 3; ++j) {
		for (int k = 0; k < 3; ++k) {
			res[j] += diff[k] * g_inv_covs[classInd][k][j];
		}
	}

	double answer = 0.0;

	for (int k = 0; k < 3; ++k) {
		answer += res[k] * diff[k];
	}
	answer *= -1.0;

	return answer;
}


__global__ void kernel(uchar4* pixels, int size, int PGP_NC) {

	int idX = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetX = gridDim.x * blockDim.x;

	for (int i = idX; i < size; i = i + offsetX) {
		uchar4* pixel = &pixels[i];
		double maxVal = classifierFunc(pixel, 0);
		int maxInd = 0;

		for (int c = 1; c < PGP_NC; ++c){

			double tmpVal = classifierFunc(pixel, c);

			if (tmpVal > maxVal){
				maxVal = tmpVal;
				maxInd = c;
			}
		}

		pixel->w = (unsigned char) maxInd;
	}

}

double avgs[32][3];
double covs[32][3][3];
double covs_inv[32][3][3];


int main(){

	
    int w, h;
    cin >> w >> h;
    uchar4* src  = (uchar4 *) malloc(sizeof(uchar4) * w * h);
    for(int i = 0; i < w * h; ++i){
        uchar4 res;
        res.x = rand() % 256;
        res.y = rand() % 256;
        res.z = rand() % 256;
        res.w = 0;
        src[i] = res;
        //cout << src[i].x << " " <<  src[i].y << " " << src[i].z << " " << src[i].w << "\n";
    }

    int nc;
	cin >> nc;
	if( nc > 32 ){
        cout << "Why nc is greater than 32?\n";
        abort();
	}

	vector< vector< pair<int, int> > > groups(nc);


	for (int i = 0; i < 32; ++i) {
		for (int j = 0; j < 3; ++j) {
			avgs[i][j] = 0.0;
		}
	}

	for (int i = 0; i < 32; ++i) {
		for (int j = 0; j < 3; ++j) {
			for (int k = 0; k < 3; ++k) {
				covs[i][j][k] = 0.0;
				covs_inv[i][j][k] = 0.0;
			}
		}
	}

    set< pair<int, int> > used;
	for (int i = 0; i < nc; ++i){

		int np;
        np = 5;
		//cin >> np;



		if( np <= 1){
            cout << "Division by 0. np should be > 1\n";
            abort();
		}
		if( np >= w * h ){
            cout << "Too much points in class!!!\n";
            abort();
		}
		if( np + used.size() >= w * h ){
            cout << "Too much points at all!!!\n";
            abort();
		}
		groups[i].resize(np);

		for (int j = 0; j < np; ++j){
                int x = rand() % w;
                int y = rand() % h;
                while( used.find(make_pair(x, y)) != used.end() ){
                    x = rand() % w;
                    y = rand() % h;
                }
                used.insert(make_pair(x, y));
                groups[i][j] = make_pair(x, y);
		}
	}

	for (int i = 0; i < nc; ++i) {

		for (int j = 0; j < groups[i].size(); ++j){
			uchar4 pixel = src[groups[i][j].first + w * groups[i][j].second];
			avgs[i][0] += pixel.x;
			avgs[i][1] += pixel.y;
			avgs[i][2] += pixel.z;
		}

		avgs[i][0] /= (double) groups[i].size();
		avgs[i][1] /= (double) groups[i].size();
		avgs[i][2] /= (double) groups[i].size();

		for (int j = 0; j < groups[i].size(); ++j){

			uchar4 pixel = src[groups[i][j].first + w * groups[i][j].second];
			double diff[3];
			double productT[3][3];

			diff[0] = (double) pixel.x - avgs[i][0];
			diff[1] = (double) pixel.y - avgs[i][1];
			diff[2] = (double) pixel.z - avgs[i][2];

			for (int s1 = 0; s1 < 3; ++s1) {
				for (int s2 = 0; s2 < 3; ++s2) {
					productT[s1][s2] = diff[s1] * diff[s2];
				}
			}

			for (int s1 = 0; s1 < 3; ++s1) {
				for (int s2 = 0; s2 < 3; ++s2) {
					covs[i][s1][s2] += productT[s1][s2];
				}
			}

		}


		double tmp = (double) groups[i].size() - 1.0;
		for (int s1 = 0; s1 < 3; ++s1) {
			for (int s2 = 0; s2 < 3; ++s2) {
				covs[i][s1][s2] /= tmp;
			}
		}

		double det = 0.0;
		det += covs[i][0][0] * covs[i][1][1] * covs[i][2][2];
		det += covs[i][0][1] * covs[i][1][2] * covs[i][2][0];
		det += covs[i][1][0] * covs[i][2][1] * covs[i][0][2];
		det -= covs[i][0][2] * covs[i][1][1] * covs[i][2][0];
		det -= covs[i][0][0] * covs[i][1][2] * covs[i][2][1];
		det -= covs[i][0][1] * covs[i][1][0] * covs[i][2][2];


		covs_inv[i][0][0] = (covs[i][1][1] * covs[i][2][2] - covs[i][2][1] * covs[i][1][2]) / det;
		covs_inv[i][0][1] = (covs[i][0][2] * covs[i][2][1] - covs[i][0][1] * covs[i][2][2]) / det;
		covs_inv[i][0][2] = (covs[i][0][1] * covs[i][1][2] - covs[i][0][2] * covs[i][1][1]) / det;
		covs_inv[i][1][0] = (covs[i][1][2] * covs[i][2][0] - covs[i][1][0] * covs[i][2][2]) / det;
		covs_inv[i][1][1] = (covs[i][0][0] * covs[i][2][2] - covs[i][0][2] * covs[i][2][0]) / det;
		covs_inv[i][1][2] = (covs[i][1][0] * covs[i][0][2] - covs[i][0][0] * covs[i][1][2]) / det;
		covs_inv[i][2][0] = (covs[i][1][0] * covs[i][2][1] - covs[i][2][0] * covs[i][1][1]) / det;
		covs_inv[i][2][1] = (covs[i][2][0] * covs[i][0][1] - covs[i][0][0] * covs[i][2][1]) / det;
		covs_inv[i][2][2] = (covs[i][0][0] * covs[i][1][1] - covs[i][1][0] * covs[i][0][1]) / det;

	}

	CSC(hipMemcpyToSymbol(HIP_SYMBOL(g_avgs), avgs, sizeof(double) * 32 * 3));
	CSC(hipMemcpyToSymbol(HIP_SYMBOL(g_inv_covs), covs_inv, sizeof(double) * 32 * 9));


	uchar4* result;
	CSC(hipMalloc( (void **) &result, w * h * sizeof(uchar4)));
	CSC(hipMemcpy(result, src, w * h * sizeof(uchar4), hipMemcpyHostToDevice));

    int sz = w * h;
    
    hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	kernel <<< 256, 256 >>> (result, sz, nc);
    CSC(hipGetLastError());
    
    CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));

	printf("time = %f\n", t);

	CSC(hipMemcpy(src, result, w * h * sizeof(uchar4), hipMemcpyDeviceToHost));
	CSC(hipFree(result));

	free(src);

	return 0;
}
